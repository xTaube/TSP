
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <random>
#include <atomic>
#include <chrono>
#include <cassert>
using namespace std;

#define MAX_X 1000
#define MAX_Y 1000
#define MAX_NO_POINTS MAX_X * MAX_Y
#define MAX_DIST MAX_X*MAX_X + MAX_Y*MAX_Y + 1
#define THREAD_SIZE 512
#define SHMEM_SIZE 512

struct city {
    int posX;
    int posY;
    bool visited;
};


__global__ void calculate_dist(city *cities, double *dist, long long int starting_point, long long n) {
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ city sh_cities[SHMEM_SIZE];
    __shared__ city sh_starting_city[SHMEM_SIZE];
    double temp;
    if (tid < n) {
        if (tid == 0) cities[starting_point].visited = true;
        sh_cities[threadIdx.x] = cities[tid];
        sh_starting_city[threadIdx.x] = cities[starting_point];
        __syncthreads();
        if (!sh_cities[threadIdx.x].visited) {
            double x = (double)sh_cities[threadIdx.x].posX-(double)sh_starting_city[threadIdx.x].posX;
            double y = (double)sh_cities[threadIdx.x].posY-(double)sh_starting_city[threadIdx.x].posY;
            temp = sqrt(x*x + y*y);
            dist[tid] = temp;
            __syncthreads();
        } else dist[tid] = MAX_DIST;
    }
}

__global__ void find_min_reduction(double *dist, double *dist_r, long long int n) {
    __shared__ double sh_dist[SHMEM_SIZE];
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        sh_dist[threadIdx.x] = dist[tid];
    else sh_dist[threadIdx.x] = MAX_DIST;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (threadIdx.x < s && sh_dist[threadIdx.x + s] > 0)
            if (sh_dist[threadIdx.x] > sh_dist[threadIdx.x + s])
                sh_dist[threadIdx.x] = sh_dist[threadIdx.x + s];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        dist_r[blockIdx.x] = sh_dist[0];
}


__global__ void sum_reduce(double *dist_vec, double *dist_sum_r, const long long int n) {
    __shared__ double partial_sum[SHMEM_SIZE];
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        partial_sum[threadIdx.x] = dist_vec[tid];
    else partial_sum[threadIdx.x] = 0;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (threadIdx.x < s)
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        dist_sum_r[blockIdx.x] = partial_sum[0];
}


vector<city> generate_cities(long long int no_cities) {
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> distribution(1, MAX_X);
    vector<city> cities;
    for (long long int i = 0; i < no_cities; i++) {
        bool unique_city;
        city c{};
        do {
            unique_city = true;
            c.posX = (int)ceil(distribution(gen));
            c.posY = (int)ceil(distribution(gen));
            for (auto &e : cities) {
                if (e.posX == c.posX && e.posY == c.posY) unique_city = false;
            }
        } while (!unique_city);
        c.visited = false;
        cities.push_back(c);
    }
    return cities;
}


double distance(city cityA, city cityB) {
    return sqrt(pow(cityA.posX - cityB.posX, 2) + pow(cityA.posY - cityB.posY, 2));
}


double total_distance(vector<city> vec) {
    double total_dist = 0;
    for (long long int i = 0; i < vec.size() - 1; i++)
        total_dist += distance(vec[i], vec[i + 1]);

    return total_dist;
}


float nn_algorithm_gpu(vector<city> vec, long long int starting_point) {
    int n = vec.size();
    size_t cities_bytes = n * sizeof(city);
    size_t dist_bytes = n * sizeof(double);

    vector<city> sorted_cities;
    vector<double> h_dist(n);
    vector<double> h_dist_r(n);
    vector<double> h_min_dists;
    vector<double> h_total_dist(n);
    city *d_cities;
    double *d_dist;
    double *d_dist_r;

    hipMalloc(&d_cities, cities_bytes);
    hipMalloc(&d_dist, dist_bytes);
    hipMalloc(&d_dist_r, dist_bytes);

    hipMemcpy(d_cities, vec.data(), cities_bytes, hipMemcpyHostToDevice);

    int threads = THREAD_SIZE;
    int blocks = (int) ceil((float) n / (float) threads);
    auto current_index = starting_point;
    sorted_cities.push_back(vec[current_index]);

    while (sorted_cities.size() != vec.size()) {
        calculate_dist<<<blocks, threads>>>(d_cities, d_dist, current_index, n);
        find_min_reduction<<<blocks, threads>>>(d_dist, d_dist_r, n);
        find_min_reduction<<<1, threads>>>(d_dist_r, d_dist_r, blocks);

        hipMemcpy(h_dist.data(), d_dist, dist_bytes, hipMemcpyDeviceToHost);
        hipMemcpy(h_dist_r.data(), d_dist_r, dist_bytes, hipMemcpyDeviceToHost);
        h_min_dists.push_back(h_dist_r[0]);
        auto it = find(h_dist.begin(), h_dist.end(), h_dist_r[0]);
        current_index = it - h_dist.begin();
        sorted_cities.push_back(vec[current_index]);
    }

    hipFree(d_cities);
    hipFree(d_dist);
    hipFree(d_dist_r);

    sorted_cities.push_back(sorted_cities[0]);
    h_min_dists.push_back(distance(sorted_cities[n - 1], sorted_cities[0]));
    double *d_min_dists;
    double *d_min_dist_sum;
    hipMalloc(&d_min_dists, dist_bytes);
    hipMalloc(&d_min_dist_sum, dist_bytes);

    hipMemcpy(d_min_dists, h_min_dists.data(), dist_bytes, hipMemcpyHostToDevice);

    sum_reduce<<<blocks, threads>>>(d_min_dists, d_min_dist_sum, n);
    sum_reduce<<<1, threads>>>(d_min_dist_sum, d_min_dist_sum, n);

    hipMemcpy(h_total_dist.data(), d_min_dist_sum, dist_bytes, hipMemcpyDeviceToHost);

    hipFree(d_min_dist_sum);
    hipFree(d_min_dists);

    return h_total_dist[0];
}


double nn_algorithm_cpu(vector<city> vec, long long int starting_point) {
    vector<city> result_vec;
    double max_dist = sqrt(pow(MAX_X, 2) + pow(MAX_Y, 2));
    auto current_index = starting_point;
    vec[current_index].visited = true;
    result_vec.push_back(vec[current_index]);
    while (result_vec.size() != vec.size()) {
        double min_dist = max_dist;
        long long int min_dist_index;
        for (long long int i = 0; i < vec.size(); i++) {
            if (i == current_index || vec[i].visited)
                continue;
            auto dist = distance(vec[current_index], vec[i]);
            if (dist < min_dist) {
                min_dist = dist;
                min_dist_index = i;
            }
        }
        current_index = min_dist_index;
        vec[current_index].visited = true;
        result_vec.push_back(vec[current_index]);
    }
    result_vec.push_back(result_vec[0]);
    return total_distance(result_vec);
}


int main() {
    long long int n;
    hipEvent_t start_gpu;
    hipEvent_t end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);
    cout << "n: " << endl;
    cin >> n;
    auto cities = generate_cities(n);
    cout << "wygenerowano" << endl;

    auto start = chrono::high_resolution_clock::now();
    auto total_dist_cpu = nn_algorithm_cpu(cities, 3);
    auto end = chrono::high_resolution_clock::now();
    auto cpu_tps_time = 0.000001*chrono::duration_cast<chrono::nanoseconds>(end - start).count();

    cout << "total distance " << total_dist_cpu << endl;
    cout << "CPU runtime: " << cpu_tps_time/1000 << "seconds"<< endl;

    hipEventRecord(start_gpu);
    auto total_dist_gpu = nn_algorithm_gpu(cities, 3);
    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);

    float sorting_ms = 0;
    hipEventElapsedTime(&sorting_ms, start_gpu, end_gpu);
    double gpu_tps_time = sorting_ms;

    cout << "total distance gpu: " << total_dist_gpu << endl;
    cout << "GPU runtime: " << gpu_tps_time/1000 << " seconds"<< endl;

    cout << "GPU did runtime test: " << cpu_tps_time/gpu_tps_time << " faster" << endl;
    return 0;
}
