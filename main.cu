
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>

using namespace std;

#define MAX_X 10
#define MAX_Y 10
#define MAX_DIST MAX_X*MAX_X + MAX_Y*MAX_Y + 1
#define THREAD_SIZE 512
#define SHMEM_SIZE 512

struct city{
    float posX;
    float posY;
    bool visited;
};


__global__ void calculate_dist(city* cities, float* dist, unsigned short starting_point, unsigned short n){
    unsigned short tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < n) {
        if (tid == 0) cities[starting_point].visited = true;
        city starting_city = cities[starting_point];
        if (!cities[tid].visited && tid != starting_point) {
            dist[tid] = sqrt(
                    pow(cities[tid].posX - starting_city.posX, 2) + pow(starting_city.posY - cities[tid].posY, 2)
                    );
        } else dist[tid] = MAX_DIST;
    }
}

__global__ void find_min_reduction(float* dist, float* dist_r, unsigned short n){
    __shared__ float sh_dist[SHMEM_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        sh_dist[threadIdx.x] = dist[tid];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (threadIdx.x < s && sh_dist[threadIdx.x+s] > 0)
            if(sh_dist[threadIdx.x] > sh_dist[threadIdx.x + s])
                sh_dist[threadIdx.x] = sh_dist[threadIdx.x +s];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        dist_r[blockIdx.x] = sh_dist[0];
}


city random_city(){
    city rand_city{};
    rand_city.posX = rand()%MAX_X;
    rand_city.posY = rand()%MAX_Y;
    rand_city.visited = false;
    return rand_city;
}


vector<city> generate_cities(unsigned short no_cities){
    vector<city> cities(no_cities);
    generate(cities.begin(), cities.end(), random_city);
    return cities;
}


float distance(city cityA, city cityB){
    return sqrt(pow(cityA.posX-cityB.posX, 2)+pow(cityA.posY-cityB.posY, 2));
}


float total_distance(vector<city> vec){
    float total_dist = 0;
    for(int i=0; i<vec.size()-1; i++)
        total_dist += distance(vec[i], vec[i+1]);

    return total_dist;
}


vector<city> nn_algorithm_gpu(vector<city> vec, unsigned short starting_point){
    int n = vec.size();
    size_t cities_bytes = n * sizeof(city);
    size_t dist_bytes = n * sizeof(float);
    vector<city> sorted_cities;
    vector<float> h_dist(n);
    vector<float> h_dist_r(n);
    city* d_cities;
    float* d_dist;
    float* d_dist_r;
    hipMalloc(&d_cities, cities_bytes);
    hipMalloc(&d_dist, dist_bytes);
    hipMalloc(&d_dist_r, dist_bytes);


    hipMemcpy(d_cities, vec.data(), cities_bytes, hipMemcpyHostToDevice);

    int threads = THREAD_SIZE;
    int blocks = (int)ceil((float)n/(float)threads);
    int current_index = starting_point;
    sorted_cities.push_back(vec[current_index]);

    while(sorted_cities.size()!=vec.size()) {
        calculate_dist<<<blocks, threads>>>(d_cities, d_dist, current_index, n);
        find_min_reduction<<<blocks, threads>>>(d_dist, d_dist_r, n);
        find_min_reduction<<<1, threads>>>(d_dist_r, d_dist_r, blocks);

        hipMemcpy(h_dist.data(), d_dist, dist_bytes, hipMemcpyDeviceToHost);
        hipMemcpy(h_dist_r.data(), d_dist_r, dist_bytes, hipMemcpyDeviceToHost);
        auto it = find(h_dist.begin(), h_dist.end(), h_dist_r[0]);
        current_index = it - h_dist.begin();
        sorted_cities.push_back(vec[current_index]);
    }
    sorted_cities.push_back(sorted_cities[0]);
    return sorted_cities;
}


vector<city> nn_algorithm_cpu(vector<city> vec, unsigned short starting_point){
    vector<city> result_vec;
    float max_dist = sqrt(pow(MAX_X, 2)+ pow(MAX_Y, 2));
    auto current_index = starting_point;
    vec[current_index].visited = true;
    result_vec.push_back(vec[current_index]);
    while(result_vec.size() != vec.size()){
        float min_dist = max_dist;
        unsigned short min_dist_index;
        for(int i=0; i<vec.size(); i++){
            if(i==current_index || vec[i].visited)
                continue;
            auto dist = distance(vec[current_index], vec[i]);
            if(dist < min_dist){
                min_dist = dist;
                min_dist_index = i;
            }
        }
        current_index = min_dist_index;
        vec[current_index].visited = true;
        result_vec.push_back(vec[current_index]);
    }
    result_vec.push_back(result_vec[0]);
    return result_vec;
}


int main() {
    unsigned short n = 5;
    auto cities = generate_cities(n);
    auto sorted_cites = nn_algorithm_cpu(cities, 2);
    cout << "total distance " << total_distance(sorted_cites) << endl;
    auto v = nn_algorithm_gpu(cities, 2);
    cout << "total distance gpu: " << total_distance(v) << endl;

    return 0;
}
